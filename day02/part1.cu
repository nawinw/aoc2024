#include "hip/hip_runtime.h"
#include <cuda/std/cmath>
#include <cassert>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>
#include "utility.h"

template<typename T>
__global__ void checker(T* data, size_t rows, size_t cols, size_t stride)
{
  size_t j = blockIdx.x * blockDim.x + threadIdx.x;
  T increasing = 1;
  T decreasing = 1;
  T has_small_diffs = 1;
  if (j < cols)
  {
    T prev = data[j];
    if (prev != -1)
    {
      size_t i = 1;
      while(data[i * stride + j] != -1)
      {
        T cur = data[i * stride + j];
        has_small_diffs = has_small_diffs & (cuda::std::abs(prev - cur) <= 3);
        increasing = increasing & (cur > prev);
        decreasing = decreasing & (cur < prev);
        prev = cur;
        i++;
      }
    }
    data[j] = (increasing ^ decreasing) & has_small_diffs;
  }
}

int main()
{
  const size_t rows = 256;
  const size_t cols = 1024;
  std::vector<int> data;
  data.resize(rows * cols);
  size_t num_cols = 0;

  std::ifstream ifs("../day02/part1.txt");
  while(ifs.good())
  {
    std::string line;
    std::getline(ifs, line);
    std::string token;

    auto ss = std::stringstream(line);
    size_t i = 0;
    while(std::getline(ss, token, ' '))
    {
      data[i++ * cols + num_cols] = std::stoi(token);
      assert(i < rows);
    }
    data[i * cols + num_cols++] = -1;
    assert(num_cols <= cols);
  }

  int *data_gpu{nullptr};
  CHECK(hipMalloc(&data_gpu, data.size() * sizeof(data[0])));
  CHECK(hipMemcpy(data_gpu, data.data(), data.size() * sizeof(data[0]), hipMemcpyDefault));

  int NUM_THREADS = 16;
  dim3 num_blocks(num_cols / NUM_THREADS + 1);
  checker<<<NUM_THREADS, num_blocks>>>(data_gpu, rows, num_cols, cols);
  CHECK(hipGetLastError());

  CHECK(hipMemcpy(data.data(), data_gpu, data.size() * sizeof(data[0]), hipMemcpyDefault));

  // for (size_t i = 0; i < rows; i++)
  // {
  //   for (size_t j = 0; j < num_cols; j++)
  //   {
  //     std::cout << data[i * cols + j] << ", ";
  //   }
  //   std::cout << std::endl;
  // }

  int sum = 0;
  for (size_t j = 0; j < num_cols; j++)
    sum += data[j];

  std::cout << "Answer: " << sum << std::endl;
}

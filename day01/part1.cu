#include "hip/hip_runtime.h"
#include <cuda/std/utility>
#include <cuda/std/cmath>
#include <fstream>
#include <iostream>
#include <vector>
#include "utility.h"

template<typename T>
__global__ void quicksort(T* start, size_t len)
{
  T pivot = start[len - 1];
  size_t low = 0;
  for (size_t i = 0; i < len; i++)
  {
      if (start[i] <= pivot)
        cuda::std::swap(start[low++], start[i]);
  }

  if (low >= 2)
  {
    hipStream_t s;
    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
    quicksort<<<1, 1, 0, s>>>(start, low - 1);
    hipStreamDestroy(s);
  }

  if (len - low > 1)
  {
    hipStream_t s;
    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
    quicksort<<<1, 1, 0, s>>>(start + low, len - low);
    hipStreamDestroy(s);
  }
}

template<typename T>
__global__ void dist(T* a, T* b, size_t len)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len)
    a[i] = cuda::std::abs(a[i] - b[i]);
}

int main()
{
  std::vector<int> list1, list2;
  std::ifstream ifs("../day01/part1.txt");
  while(ifs.good())
  {
    int a, b;
    ifs >> a >> b;
    list1.push_back(a);
    list2.push_back(b);
  }

  int *list1_gpu{nullptr};
  int *list2_gpu{nullptr};
  CHECK(hipMalloc(&list1_gpu, list1.size() * sizeof(list1[0])));
  CHECK(hipMalloc(&list2_gpu, list2.size() * sizeof(list2[0])));
  CHECK(hipMemcpy(list1_gpu, list1.data(), list1.size() * sizeof(list1[0]), hipMemcpyDefault));
  CHECK(hipMemcpy(list2_gpu, list2.data(), list2.size() * sizeof(list2[0]), hipMemcpyDefault));

  quicksort<<< 1, 1 >>>(list1_gpu, list1.size());
  CHECK(hipGetLastError());

  quicksort<<< 1, 1 >>>(list2_gpu, list2.size());
  CHECK(hipGetLastError());

  int NUM_THREADS = 16;
  dim3 NUM_BLOCKS(list1.size() / NUM_THREADS + 1);
  dist<<< NUM_THREADS, NUM_BLOCKS >>>(list1_gpu, list2_gpu, list1.size());
  CHECK(hipGetLastError());

  CHECK(hipMemcpy(list1.data(), list1_gpu, list1.size() * sizeof(list1[0]), hipMemcpyDefault));
  CHECK(hipMemcpy(list2.data(), list2_gpu, list2.size() * sizeof(list2[0]), hipMemcpyDefault));

  uint64_t sum = 0;
  for (const auto& item: list1)
    sum += item;
  std::cout << "Answer: " << sum << std::endl;
}